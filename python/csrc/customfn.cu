#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <flashinfer/div_clamp_to.cuh>

#include "flashinfer_ops.h"
#include "pytorch_extension_utils.h"

using namespace flashinfer;

void div_clamp_to(torch::Tensor& output, 
                  const torch::Tensor& input,
                  const torch::Tensor& scale) {
    int hidden_size = input.size(-1);
    int64_t num_tokens = input.numel() / input.size(-1);
    dim3 grid(num_tokens);
    const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input.scalar_type(), input_type, [&] {
        DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP8(output.scalar_type(), output_type, [&] {
            uint32_t vec_size = 16 / sizeof(input_type);
            dim3 block(std::min(hidden_size / vec_size, 1024U));

            flashinfer::customfn::div_clamp_to<output_type, input_type><<<grid, block, 0, stream>>>(
                static_cast<output_type*>(output.data_ptr()),
                static_cast<input_type*>(input.data_ptr()), 
                static_cast<input_type*>(scale.data_ptr()),
                hidden_size
            );
            return true;
        });
        return true;     
    });
}