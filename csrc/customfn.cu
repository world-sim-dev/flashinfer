#include "hip/hip_runtime.h"
#include <flashinfer/div_clamp_to.cuh>

#include "pytorch_extension_utils.h"

using namespace flashinfer;

void div_clamp_to(at::Tensor& output, 
                  const at::Tensor& input, 
                  const at::Tensor& scale, 
                  int64_t hip_stream) {
  TORCH_CHECK(output.is_cuda(), "output must be a CUDA tensor");
  TORCH_CHECK(input.is_cuda(), "input must be a CUDA tensor");
  TORCH_CHECK(scale.is_cuda(), "scale must be a CUDA tensor");
  TORCH_CHECK(scale.dim() == 1, "Expected 1D tensor for scale");
  TORCH_CHECK(input.size(-1) == scale.size(0), "hidden_size between input and scale must match");
  TORCH_CHECK(is_float8_tensor(output), "output must be Float8_e4m3 or Float8_e5m2");
  TORCH_CHECK(input.scalar_type() == at::ScalarType::BFloat16 || input.scalar_type() == at::ScalarType::Half,
              "input must be BFloat16 or Half");
  TORCH_CHECK(scale.scalar_type() == at::ScalarType::Float,
              "scale must be Float");

  int hidden_size = input.size(-1);
  int64_t num_tokens = input.numel() / input.size(-1);
  dim3 grid(num_tokens);

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input.scalar_type(), input_type, [&] {
    DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP8(output.scalar_type(), output_type, [&] {
      uint32_t vec_size = 16 / sizeof(input_type);
      dim3 block(std::min(hidden_size / vec_size, 1024U));

      flashinfer::customfn::div_clamp_to<output_type, input_type><<<grid, block, 0, stream>>>(
          static_cast<output_type*>(output.data_ptr()),
          static_cast<input_type*>(input.data_ptr()), 
          static_cast<float*>(scale.data_ptr()),
          hidden_size
      );
      return true;
    });
    return true;     
  });
}
